#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/gpu/device/common.hpp"
#include "opencv2/gpu/device/utility.hpp"
#include "opencv2/gpu/device/functional.hpp"
#include "opencv2/gpu/device/limits.hpp"
#include "opencv2/gpu/device/vec_math.hpp"
#include "opencv2/gpu/device/reduce.hpp"

#include "opencv2/core/core.hpp"

#include "cvconfig.h"

using namespace cv::gpu;
using namespace cv::gpu::device;

namespace pyrlk
{
    __constant__ int c_winSize_x;
    __constant__ int c_winSize_y;
    __constant__ int c_halfWin_x;
    __constant__ int c_halfWin_y;
    __constant__ int c_iters;

#define CUDA_CONSTANTS(index) \
    __constant__ int c_winSize_x##index; \
    __constant__ int c_winSize_y##index; \
    __constant__ int c_halfWin_x##index; \
    __constant__ int c_halfWin_y##index; \
    __constant__ int c_iters##index;

    CUDA_CONSTANTS(0)
    CUDA_CONSTANTS(1)
    CUDA_CONSTANTS(2)
    CUDA_CONSTANTS(3)
    CUDA_CONSTANTS(4)

    template <int index> struct c_multi_winSize_x;
    template <int index> struct c_multi_winSize_y;
    template <int index> struct c_multi_halfWin_x;
    template <int index> struct c_multi_halfWin_y;
    template <int index> struct c_multi_iters;

#define CUDA_CONSTANTS_ACCESSOR(index) \
    template <> struct c_multi_winSize_x<index> \
    { static __device__ __forceinline__ int get(void){ return c_winSize_x##index;} }; \
    template <> struct c_multi_winSize_y<index> \
    { static __device__ __forceinline__ int get(void){ return c_winSize_y##index;} }; \
    template <> struct c_multi_halfWin_x<index> \
    { static __device__ __forceinline__ int get(void){ return c_halfWin_x##index;} }; \
    template <> struct c_multi_halfWin_y<index> \
    { static __device__ __forceinline__ int get(void){ return c_halfWin_y##index;} }; \
    template <> struct c_multi_iters<index> \
    { static __device__ __forceinline__ int get(void){ return c_iters##index;} };

    CUDA_CONSTANTS_ACCESSOR(0)
    CUDA_CONSTANTS_ACCESSOR(1)
    CUDA_CONSTANTS_ACCESSOR(2)
    CUDA_CONSTANTS_ACCESSOR(3)
    CUDA_CONSTANTS_ACCESSOR(4)

    texture<float, hipTextureType2D, hipReadModeElementType>
            tex_If(false, hipFilterModeLinear, hipAddressModeClamp);
    texture<float4, hipTextureType2D, hipReadModeElementType>
            tex_If4(false, hipFilterModeLinear, hipAddressModeClamp);
    texture<uchar, hipTextureType2D, hipReadModeElementType>
            tex_Ib(false, hipFilterModePoint, hipAddressModeClamp);

    texture<float, hipTextureType2D, hipReadModeElementType>
            tex_Jf(false, hipFilterModeLinear, hipAddressModeClamp);
    texture<float4, hipTextureType2D, hipReadModeElementType>
            tex_Jf4(false, hipFilterModeLinear, hipAddressModeClamp);

    template <int cn> struct Tex_I;
    template <> struct Tex_I<1>
    {
        static __device__ __forceinline__ float read(float x, float y)
        {
            return tex2D(tex_If, x, y);
        }
    };
    template <> struct Tex_I<4>
    {
        static __device__ __forceinline__ float4 read(float x, float y)
        {
            return tex2D(tex_If4, x, y);
        }
    };

    template <int cn> struct Tex_J;
    template <> struct Tex_J<1>
    {
        static __device__ __forceinline__ float read(float x, float y)
        {
            return tex2D(tex_Jf, x, y);
        }
    };
    template <> struct Tex_J<4>
    {
        static __device__ __forceinline__ float4 read(float x, float y)
        {
            return tex2D(tex_Jf4, x, y);
        }
    };

    //--------------------------------------------------------------------------

#define CUDA_DECL_TEX_MULTI(texname, type, filtermode) \
    texture<type, hipTextureType2D, hipReadModeElementType> \
            texname##_multi0(false, filtermode, hipAddressModeClamp); \
    texture<type, hipTextureType2D, hipReadModeElementType> \
            texname##_multi1(false, filtermode, hipAddressModeClamp); \
    texture<type, hipTextureType2D, hipReadModeElementType> \
            texname##_multi2(false, filtermode, hipAddressModeClamp); \
    texture<type, hipTextureType2D, hipReadModeElementType> \
            texname##_multi3(false, filtermode, hipAddressModeClamp); \
    texture<type, hipTextureType2D, hipReadModeElementType> \
            texname##_multi4(false, filtermode, hipAddressModeClamp); \

    CUDA_DECL_TEX_MULTI(tex_If1, float, hipFilterModeLinear)
    CUDA_DECL_TEX_MULTI(tex_If4, float4, hipFilterModeLinear)
    CUDA_DECL_TEX_MULTI(tex_Ib1, uchar, hipFilterModePoint)
    CUDA_DECL_TEX_MULTI(tex_Jf1, float, hipFilterModeLinear)
    CUDA_DECL_TEX_MULTI(tex_Jf4, float4, hipFilterModeLinear)

    template <int cn, int index> struct Tex_I_multi;
    template <int cn, int index> struct Tex_J_multi;
    template <int cn, int index> struct Tex_B_multi;

#define CUDA_DECL_TEX_MULTI_ACCESS(accessorname, texname, cn, returntype) \
    template <> struct accessorname##_multi<cn, 0> \
    { static __device__ __forceinline__ returntype read(float x, float y) \
        { return tex2D(texname##cn##_multi0, x, y); } }; \
    template <> struct accessorname##_multi<cn, 1> \
    { static __device__ __forceinline__ returntype read(float x, float y) \
        { return tex2D(texname##cn##_multi1, x, y); } }; \
    template <> struct accessorname##_multi<cn, 2> \
    { static __device__ __forceinline__ returntype read(float x, float y) \
        { return tex2D(texname##cn##_multi2, x, y); } }; \
    template <> struct accessorname##_multi<cn, 3> \
    { static __device__ __forceinline__ returntype read(float x, float y) \
        { return tex2D(texname##cn##_multi3, x, y); } }; \
    template <> struct accessorname##_multi<cn, 4> \
    { static __device__ __forceinline__ returntype read(float x, float y) \
        { return tex2D(texname##cn##_multi4, x, y); } };

    CUDA_DECL_TEX_MULTI_ACCESS(Tex_I, tex_If, 1, float)
    CUDA_DECL_TEX_MULTI_ACCESS(Tex_I, tex_If, 4, float4)

    CUDA_DECL_TEX_MULTI_ACCESS(Tex_B, tex_Ib, 1, uchar)

    CUDA_DECL_TEX_MULTI_ACCESS(Tex_J, tex_Jf, 1, float)
    CUDA_DECL_TEX_MULTI_ACCESS(Tex_J, tex_Jf, 4, float4)

    //--------------------------------------------------------------------------

    __device__ __forceinline__ void accum(float& dst, float val)
    {
        dst += val;
    }
    __device__ __forceinline__ void accum(float& dst, const float4& val)
    {
        dst += val.x + val.y + val.z;
    }

    __device__ __forceinline__ float abs_(float a)
    {
        return ::fabsf(a);
    }
    __device__ __forceinline__ float4 abs_(const float4& a)
    {
        return abs(a);
    }

    template <int cn, int PATCH_X, int PATCH_Y, bool calcErr>
    __global__ void sparseKernel(const float2* prevPts, float2* nextPts, uchar* status, float* err, const int level, const int rows, const int cols)
    {
    #if __CUDA_ARCH__ <= 110
        const int BLOCK_SIZE = 128;
    #else
        const int BLOCK_SIZE = 256;
    #endif

        __shared__ float smem1[BLOCK_SIZE];
        __shared__ float smem2[BLOCK_SIZE];
        __shared__ float smem3[BLOCK_SIZE];

        const unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

        float2 prevPt = prevPts[blockIdx.x];
        prevPt.x *= (1.0f / (1 << level));
        prevPt.y *= (1.0f / (1 << level));

        if (prevPt.x < 0 || prevPt.x >= cols || prevPt.y < 0 || prevPt.y >= rows)
        {
            if (tid == 0 && level == 0)
                status[blockIdx.x] = 0;

            return;
        }

        prevPt.x -= c_halfWin_x;
        prevPt.y -= c_halfWin_y;

        // extract the patch from the first image, compute covariation matrix of derivatives

        float A11 = 0;
        float A12 = 0;
        float A22 = 0;

        typedef typename TypeVec<float, cn>::vec_type work_type;

        work_type I_patch   [PATCH_Y][PATCH_X];
        work_type dIdx_patch[PATCH_Y][PATCH_X];
        work_type dIdy_patch[PATCH_Y][PATCH_X];

        for (int yBase = threadIdx.y, i = 0; yBase < c_winSize_y; yBase += blockDim.y, ++i)
        {
            for (int xBase = threadIdx.x, j = 0; xBase < c_winSize_x; xBase += blockDim.x, ++j)
            {
                float x = prevPt.x + xBase + 0.5f;
                float y = prevPt.y + yBase + 0.5f;

                I_patch[i][j] = Tex_I<cn>::read(x, y);

                // Sharr Deriv

                work_type dIdx = 3.0f * Tex_I<cn>::read(x+1, y-1) + 10.0f * Tex_I<cn>::read(x+1, y) + 3.0f * Tex_I<cn>::read(x+1, y+1) -
                                 (3.0f * Tex_I<cn>::read(x-1, y-1) + 10.0f * Tex_I<cn>::read(x-1, y) + 3.0f * Tex_I<cn>::read(x-1, y+1));

                work_type dIdy = 3.0f * Tex_I<cn>::read(x-1, y+1) + 10.0f * Tex_I<cn>::read(x, y+1) + 3.0f * Tex_I<cn>::read(x+1, y+1) -
                                (3.0f * Tex_I<cn>::read(x-1, y-1) + 10.0f * Tex_I<cn>::read(x, y-1) + 3.0f * Tex_I<cn>::read(x+1, y-1));

                dIdx_patch[i][j] = dIdx;
                dIdy_patch[i][j] = dIdy;

                accum(A11, dIdx * dIdx);
                accum(A12, dIdx * dIdy);
                accum(A22, dIdy * dIdy);
            }
        }

        reduce<BLOCK_SIZE>(smem_tuple(smem1, smem2, smem3), thrust::tie(A11, A12, A22), tid, thrust::make_tuple(plus<float>(), plus<float>(), plus<float>()));

    #if __CUDA_ARCH__ >= 300
        if (tid == 0)
        {
            smem1[0] = A11;
            smem2[0] = A12;
            smem3[0] = A22;
        }
    #endif

        __syncthreads();

        A11 = smem1[0];
        A12 = smem2[0];
        A22 = smem3[0];

        float D = A11 * A22 - A12 * A12;

        if (D < numeric_limits<float>::epsilon())
        {
            if (tid == 0 && level == 0)
                status[blockIdx.x] = 0;

            return;
        }

        D = 1.f / D;

        A11 *= D;
        A12 *= D;
        A22 *= D;

        float2 nextPt = nextPts[blockIdx.x];
        nextPt.x *= 2.f;
        nextPt.y *= 2.f;

        nextPt.x -= c_halfWin_x;
        nextPt.y -= c_halfWin_y;

        for (int k = 0; k < c_iters; ++k)
        {
            if (nextPt.x < -c_halfWin_x || nextPt.x >= cols || nextPt.y < -c_halfWin_y || nextPt.y >= rows)
            {
                if (tid == 0 && level == 0)
                    status[blockIdx.x] = 0;

                return;
            }

            float b1 = 0;
            float b2 = 0;

            for (int y = threadIdx.y, i = 0; y < c_winSize_y; y += blockDim.y, ++i)
            {
                for (int x = threadIdx.x, j = 0; x < c_winSize_x; x += blockDim.x, ++j)
                {
                    work_type I_val = I_patch[i][j];
                    work_type J_val = Tex_J<cn>::read(nextPt.x + x + 0.5f, nextPt.y + y + 0.5f);

                    work_type diff = (J_val - I_val) * 32.0f;

                    accum(b1, diff * dIdx_patch[i][j]);
                    accum(b2, diff * dIdy_patch[i][j]);
                }
            }

            reduce<BLOCK_SIZE>(smem_tuple(smem1, smem2), thrust::tie(b1, b2), tid, thrust::make_tuple(plus<float>(), plus<float>()));

        #if __CUDA_ARCH__ >= 300
            if (tid == 0)
            {
                smem1[0] = b1;
                smem2[0] = b2;
            }
        #endif

            __syncthreads();

            b1 = smem1[0];
            b2 = smem2[0];

            float2 delta;
            delta.x = A12 * b2 - A22 * b1;
            delta.y = A12 * b1 - A11 * b2;

            nextPt.x += delta.x;
            nextPt.y += delta.y;

            if (::fabs(delta.x) < 0.01f && ::fabs(delta.y) < 0.01f)
                break;
        }

        float errval = 0;
        if (calcErr)
        {
            for (int y = threadIdx.y, i = 0; y < c_winSize_y; y += blockDim.y, ++i)
            {
                for (int x = threadIdx.x, j = 0; x < c_winSize_x; x += blockDim.x, ++j)
                {
                    work_type I_val = I_patch[i][j];
                    work_type J_val = Tex_J<cn>::read(nextPt.x + x + 0.5f, nextPt.y + y + 0.5f);

                    work_type diff = J_val - I_val;

                    accum(errval, abs_(diff));
                }
            }

            reduce<BLOCK_SIZE>(smem1, errval, tid, plus<float>());
        }

        if (tid == 0)
        {
            nextPt.x += c_halfWin_x;
            nextPt.y += c_halfWin_y;

            nextPts[blockIdx.x] = nextPt;

            if (calcErr)
                err[blockIdx.x] = static_cast<float>(errval) / (cn * c_winSize_x * c_winSize_y);
        }
    }

#if defined(HAVE_TBB)
    template <int cn, int index, int PATCH_X, int PATCH_Y, bool calcErr>
    __global__ void sparseKernel_multi(const float2* prevPts, float2* nextPts, uchar* status, float* err, const int level, const int rows, const int cols)
    {
    #if __CUDA_ARCH__ <= 110
        const int BLOCK_SIZE = 128;
    #else
        const int BLOCK_SIZE = 256;
    #endif

        __shared__ float smem1[BLOCK_SIZE];
        __shared__ float smem2[BLOCK_SIZE];
        __shared__ float smem3[BLOCK_SIZE];

        const unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

        float2 prevPt = prevPts[blockIdx.x];
        prevPt.x *= (1.0f / (1 << level));
        prevPt.y *= (1.0f / (1 << level));

        if (prevPt.x < 0 || prevPt.x >= cols || prevPt.y < 0 || prevPt.y >= rows)
        {
            if (tid == 0 && level == 0)
                status[blockIdx.x] = 0;

            return;
        }

        prevPt.x -= c_multi_halfWin_x<index>::get();
        prevPt.y -= c_multi_halfWin_y<index>::get();

        // extract the patch from the first image, compute covariation matrix of derivatives

        float A11 = 0;
        float A12 = 0;
        float A22 = 0;

        typedef typename TypeVec<float, cn>::vec_type work_type;

        work_type I_patch   [PATCH_Y][PATCH_X];
        work_type dIdx_patch[PATCH_Y][PATCH_X];
        work_type dIdy_patch[PATCH_Y][PATCH_X];

        for (int yBase = threadIdx.y, i = 0; yBase < c_multi_winSize_y<index>::get(); yBase += blockDim.y, ++i)
        {
            for (int xBase = threadIdx.x, j = 0; xBase < c_multi_winSize_x<index>::get(); xBase += blockDim.x, ++j)
            {
                float x = prevPt.x + xBase + 0.5f;
                float y = prevPt.y + yBase + 0.5f;

                I_patch[i][j] = Tex_I_multi<cn,index>::read(x, y);

                // Sharr Deriv

                work_type dIdx = 3.0f * Tex_I_multi<cn,index>::read(x+1, y-1) + 10.0f * Tex_I_multi<cn,index>::read(x+1, y) + 3.0f * Tex_I_multi<cn,index>::read(x+1, y+1) -
                                 (3.0f * Tex_I_multi<cn,index>::read(x-1, y-1) + 10.0f * Tex_I_multi<cn,index>::read(x-1, y) + 3.0f * Tex_I_multi<cn,index>::read(x-1, y+1));

                work_type dIdy = 3.0f * Tex_I_multi<cn,index>::read(x-1, y+1) + 10.0f * Tex_I_multi<cn,index>::read(x, y+1) + 3.0f * Tex_I_multi<cn,index>::read(x+1, y+1) -
                                (3.0f * Tex_I_multi<cn,index>::read(x-1, y-1) + 10.0f * Tex_I_multi<cn,index>::read(x, y-1) + 3.0f * Tex_I_multi<cn,index>::read(x+1, y-1));

                dIdx_patch[i][j] = dIdx;
                dIdy_patch[i][j] = dIdy;

                accum(A11, dIdx * dIdx);
                accum(A12, dIdx * dIdy);
                accum(A22, dIdy * dIdy);
            }
        }

        reduce<BLOCK_SIZE>(smem_tuple(smem1, smem2, smem3), thrust::tie(A11, A12, A22), tid, thrust::make_tuple(plus<float>(), plus<float>(), plus<float>()));

    #if __CUDA_ARCH__ >= 300
        if (tid == 0)
        {
            smem1[0] = A11;
            smem2[0] = A12;
            smem3[0] = A22;
        }
    #endif

        __syncthreads();

        A11 = smem1[0];
        A12 = smem2[0];
        A22 = smem3[0];

        float D = A11 * A22 - A12 * A12;

        if (abs_(D) < numeric_limits<float>::epsilon())
        {
            if (tid == 0 && level == 0)
                status[blockIdx.x] = 0;

            return;
        }

        D = 1.f / D;

        A11 *= D;
        A12 *= D;
        A22 *= D;

        float2 nextPt = nextPts[blockIdx.x];
        nextPt.x *= 2.f;
        nextPt.y *= 2.f;

        nextPt.x -= c_multi_halfWin_x<index>::get();
        nextPt.y -= c_multi_halfWin_y<index>::get();

        for (int k = 0; k < c_multi_iters<index>::get(); ++k)
        {
            if (nextPt.x < -c_multi_halfWin_x<index>::get() || nextPt.x >= cols || nextPt.y < -c_multi_halfWin_y<index>::get() || nextPt.y >= rows)
            {
                if (tid == 0 && level == 0)
                    status[blockIdx.x] = 0;

                return;
            }

            float b1 = 0;
            float b2 = 0;

            for (int y = threadIdx.y, i = 0; y < c_multi_winSize_y<index>::get(); y += blockDim.y, ++i)
            {
                for (int x = threadIdx.x, j = 0; x < c_multi_winSize_x<index>::get(); x += blockDim.x, ++j)
                {
                    work_type I_val = I_patch[i][j];
                    work_type J_val = Tex_J_multi<cn,index>::read(nextPt.x + x + 0.5f, nextPt.y + y + 0.5f);

                    work_type diff = (J_val - I_val) * 32.0f;

                    accum(b1, diff * dIdx_patch[i][j]);
                    accum(b2, diff * dIdy_patch[i][j]);
                }
            }

            reduce<BLOCK_SIZE>(smem_tuple(smem1, smem2), thrust::tie(b1, b2), tid, thrust::make_tuple(plus<float>(), plus<float>()));

        #if __CUDA_ARCH__ >= 300
            if (tid == 0)
            {
                smem1[0] = b1;
                smem2[0] = b2;
            }
        #endif

            __syncthreads();

            b1 = smem1[0];
            b2 = smem2[0];

            float2 delta;
            delta.x = A12 * b2 - A22 * b1;
            delta.y = A12 * b1 - A11 * b2;

            nextPt.x += delta.x;
            nextPt.y += delta.y;

            if (::fabs(delta.x) < 0.01f && ::fabs(delta.y) < 0.01f)
                break;
        }

        float errval = 0;
        if (calcErr)
        {
            for (int y = threadIdx.y, i = 0; y < c_multi_winSize_y<index>::get(); y += blockDim.y, ++i)
            {
                for (int x = threadIdx.x, j = 0; x < c_multi_winSize_x<index>::get(); x += blockDim.x, ++j)
                {
                    work_type I_val = I_patch[i][j];
                    work_type J_val = Tex_J_multi<cn,index>::read(nextPt.x + x + 0.5f, nextPt.y + y + 0.5f);

                    work_type diff = J_val - I_val;

                    accum(errval, abs_(diff));
                }
            }

            reduce<BLOCK_SIZE>(smem1, errval, tid, plus<float>());
        }

        if (tid == 0)
        {
            nextPt.x += c_multi_halfWin_x<index>::get();
            nextPt.y += c_multi_halfWin_y<index>::get();

            nextPts[blockIdx.x] = nextPt;

            if (calcErr)
                err[blockIdx.x] = static_cast<float>(errval) / (cn * c_multi_winSize_x<index>::get() * c_multi_winSize_y<index>::get());
        }
    }
#endif // defined(HAVE_TBB)

    template <int cn, int PATCH_X, int PATCH_Y>
    void sparse_caller(int rows, int cols, const float2* prevPts, float2* nextPts, uchar* status, float* err, int ptcount,
                       int level, dim3 block, hipStream_t stream)
    {
        dim3 grid(ptcount);

        if (level == 0 && err)
            sparseKernel<cn, PATCH_X, PATCH_Y, true><<<grid, block>>>(prevPts, nextPts, status, err, level, rows, cols);
        else
            sparseKernel<cn, PATCH_X, PATCH_Y, false><<<grid, block>>>(prevPts, nextPts, status, err, level, rows, cols);

        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

#if defined(HAVE_TBB)
    template <int cn, int index, int PATCH_X, int PATCH_Y>
    void sparse_caller_multi(int rows, int cols, const float2* prevPts, float2* nextPts, uchar* status, float* err, int ptcount,
                       int level, dim3 block, hipStream_t stream)
    {
        dim3 grid(ptcount);

        if (level == 0 && err)
            sparseKernel_multi<cn, index, PATCH_X, PATCH_Y, true><<<grid, block>>>(prevPts, nextPts, status, err, level, rows, cols);
        else
            sparseKernel_multi<cn, index, PATCH_X, PATCH_Y, false><<<grid, block>>>(prevPts, nextPts, status, err, level, rows, cols);

        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

#endif // defined(HAVE_TBB)

    template <bool calcErr>
    __global__ void denseKernel(PtrStepf u, PtrStepf v, const PtrStepf prevU, const PtrStepf prevV, PtrStepf err, const int rows, const int cols)
    {
        extern __shared__ int smem[];

        const int patchWidth  = blockDim.x + 2 * c_halfWin_x;
        const int patchHeight = blockDim.y + 2 * c_halfWin_y;

        int* I_patch = smem;
        int* dIdx_patch = I_patch + patchWidth * patchHeight;
        int* dIdy_patch = dIdx_patch + patchWidth * patchHeight;

        const int xBase = blockIdx.x * blockDim.x;
        const int yBase = blockIdx.y * blockDim.y;

        for (int i = threadIdx.y; i < patchHeight; i += blockDim.y)
        {
            for (int j = threadIdx.x; j < patchWidth; j += blockDim.x)
            {
                float x = xBase - c_halfWin_x + j + 0.5f;
                float y = yBase - c_halfWin_y + i + 0.5f;

                I_patch[i * patchWidth + j] = tex2D(tex_Ib, x, y);

                // Sharr Deriv

                dIdx_patch[i * patchWidth + j] = 3 * tex2D(tex_Ib, x+1, y-1) + 10 * tex2D(tex_Ib, x+1, y) + 3 * tex2D(tex_Ib, x+1, y+1) -
                                                (3 * tex2D(tex_Ib, x-1, y-1) + 10 * tex2D(tex_Ib, x-1, y) + 3 * tex2D(tex_Ib, x-1, y+1));

                dIdy_patch[i * patchWidth + j] = 3 * tex2D(tex_Ib, x-1, y+1) + 10 * tex2D(tex_Ib, x, y+1) + 3 * tex2D(tex_Ib, x+1, y+1) -
                                                (3 * tex2D(tex_Ib, x-1, y-1) + 10 * tex2D(tex_Ib, x, y-1) + 3 * tex2D(tex_Ib, x+1, y-1));
            }
        }

        __syncthreads();

        const int x = xBase + threadIdx.x;
        const int y = yBase + threadIdx.y;

        if (x >= cols || y >= rows)
            return;

        int A11i = 0;
        int A12i = 0;
        int A22i = 0;

        for (int i = 0; i < c_winSize_y; ++i)
        {
            for (int j = 0; j < c_winSize_x; ++j)
            {
                int dIdx = dIdx_patch[(threadIdx.y + i) * patchWidth + (threadIdx.x + j)];
                int dIdy = dIdy_patch[(threadIdx.y + i) * patchWidth + (threadIdx.x + j)];

                A11i += dIdx * dIdx;
                A12i += dIdx * dIdy;
                A22i += dIdy * dIdy;
            }
        }

        float A11 = A11i;
        float A12 = A12i;
        float A22 = A22i;

        float D = A11 * A22 - A12 * A12;

        if (D < numeric_limits<float>::epsilon())
        {
            if (calcErr)
                err(y, x) = numeric_limits<float>::max();

            return;
        }

        D = 1.f / D;

        A11 *= D;
        A12 *= D;
        A22 *= D;

        float2 nextPt;
        nextPt.x = x + prevU(y/2, x/2) * 2.0f;
        nextPt.y = y + prevV(y/2, x/2) * 2.0f;

        for (int k = 0; k < c_iters; ++k)
        {
            if (nextPt.x < 0 || nextPt.x >= cols || nextPt.y < 0 || nextPt.y >= rows)
            {
                if (calcErr)
                    err(y, x) = numeric_limits<float>::max();

                return;
            }

            int b1 = 0;
            int b2 = 0;

            for (int i = 0; i < c_winSize_y; ++i)
            {
                for (int j = 0; j < c_winSize_x; ++j)
                {
                    int I = I_patch[(threadIdx.y + i) * patchWidth + threadIdx.x + j];
                    int J = tex2D(tex_Jf, nextPt.x - c_halfWin_x + j + 0.5f, nextPt.y - c_halfWin_y + i + 0.5f);

                    int diff = (J - I) * 32;

                    int dIdx = dIdx_patch[(threadIdx.y + i) * patchWidth + (threadIdx.x + j)];
                    int dIdy = dIdy_patch[(threadIdx.y + i) * patchWidth + (threadIdx.x + j)];

                    b1 += diff * dIdx;
                    b2 += diff * dIdy;
                }
            }

            float2 delta;
            delta.x = A12 * b2 - A22 * b1;
            delta.y = A12 * b1 - A11 * b2;

            nextPt.x += delta.x;
            nextPt.y += delta.y;

            if (::fabs(delta.x) < 0.01f && ::fabs(delta.y) < 0.01f)
                break;
        }

        u(y, x) = nextPt.x - x;
        v(y, x) = nextPt.y - y;

        if (calcErr)
        {
            int errval = 0;

            for (int i = 0; i < c_winSize_y; ++i)
            {
                for (int j = 0; j < c_winSize_x; ++j)
                {
                    int I = I_patch[(threadIdx.y + i) * patchWidth + threadIdx.x + j];
                    int J = tex2D(tex_Jf, nextPt.x - c_halfWin_x + j + 0.5f, nextPt.y - c_halfWin_y + i + 0.5f);

                    errval += ::abs(J - I);
                }
            }

            err(y, x) = static_cast<float>(errval) / (c_winSize_x * c_winSize_y);
        }
    }

    void loadConstants(int2 winSize, int iters)
    {
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_winSize_x), &winSize.x, sizeof(int)) );
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_winSize_y), &winSize.y, sizeof(int)) );

        int2 halfWin = make_int2((winSize.x - 1) / 2, (winSize.y - 1) / 2);
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_halfWin_x), &halfWin.x, sizeof(int)) );
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_halfWin_y), &halfWin.y, sizeof(int)) );

        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_iters), &iters, sizeof(int)) );
    }

#if defined(HAVE_TBB)
    void loadConstants_multi(int2 winSize, int iters, int index, hipStream_t stream = 0)
    {
        int2 halfWin;
#define COPY_TO_SYMBOL_CALL(index) \
        cudaSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(c_winSize_x##index), &winSize.x, sizeof(int), 0, hipMemcpyHostToDevice, stream) ); \
        cudaSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(c_winSize_y##index), &winSize.y, sizeof(int), 0, hipMemcpyHostToDevice, stream) ); \
        halfWin = make_int2((winSize.x - 1) / 2, (winSize.y - 1) / 2); \
        cudaSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(c_halfWin_x##index), &halfWin.x, sizeof(int), 0, hipMemcpyHostToDevice, stream) ); \
        cudaSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(c_halfWin_y##index), &halfWin.y, sizeof(int), 0, hipMemcpyHostToDevice, stream) ); \
        cudaSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(c_iters##index), &iters, sizeof(int), 0, hipMemcpyHostToDevice, stream) );

        switch(index)
        {
            case 0: COPY_TO_SYMBOL_CALL(0) break;
            case 1: COPY_TO_SYMBOL_CALL(1) break;
            case 2: COPY_TO_SYMBOL_CALL(2) break;
            case 3: COPY_TO_SYMBOL_CALL(3) break;
            case 4: COPY_TO_SYMBOL_CALL(4) break;
            default: CV_Error(CV_StsBadArg, "invalid execution line index"); break;
        }
    }
#endif // defined(HAVE_TBB)

    void sparse1(PtrStepSzf I, PtrStepSzf J, const float2* prevPts, float2* nextPts, uchar* status, float* err, int ptcount,
                 int level, dim3 block, dim3 patch, hipStream_t stream)
    {
        typedef void (*func_t)(int rows, int cols, const float2* prevPts, float2* nextPts, uchar* status, float* err, int ptcount,
                               int level, dim3 block, hipStream_t stream);

        static const func_t funcs[5][5] =
        {
            {sparse_caller<1, 1, 1>, sparse_caller<1, 2, 1>, sparse_caller<1, 3, 1>, sparse_caller<1, 4, 1>, sparse_caller<1, 5, 1>},
            {sparse_caller<1, 1, 2>, sparse_caller<1, 2, 2>, sparse_caller<1, 3, 2>, sparse_caller<1, 4, 2>, sparse_caller<1, 5, 2>},
            {sparse_caller<1, 1, 3>, sparse_caller<1, 2, 3>, sparse_caller<1, 3, 3>, sparse_caller<1, 4, 3>, sparse_caller<1, 5, 3>},
            {sparse_caller<1, 1, 4>, sparse_caller<1, 2, 4>, sparse_caller<1, 3, 4>, sparse_caller<1, 4, 4>, sparse_caller<1, 5, 4>},
            {sparse_caller<1, 1, 5>, sparse_caller<1, 2, 5>, sparse_caller<1, 3, 5>, sparse_caller<1, 4, 5>, sparse_caller<1, 5, 5>}
        };

        bindTexture(&tex_If, I);
        bindTexture(&tex_Jf, J);

        funcs[patch.y - 1][patch.x - 1](I.rows, I.cols, prevPts, nextPts, status, err, ptcount,
            level, block, stream);
    }

    void sparse4(PtrStepSz<float4> I, PtrStepSz<float4> J, const float2* prevPts, float2* nextPts, uchar* status, float* err, int ptcount,
                 int level, dim3 block, dim3 patch, hipStream_t stream)
    {
        typedef void (*func_t)(int rows, int cols, const float2* prevPts, float2* nextPts, uchar* status, float* err, int ptcount,
                               int level, dim3 block, hipStream_t stream);

        static const func_t funcs[5][5] =
        {
            {sparse_caller<4, 1, 1>, sparse_caller<4, 2, 1>, sparse_caller<4, 3, 1>, sparse_caller<4, 4, 1>, sparse_caller<4, 5, 1>},
            {sparse_caller<4, 1, 2>, sparse_caller<4, 2, 2>, sparse_caller<4, 3, 2>, sparse_caller<4, 4, 2>, sparse_caller<4, 5, 2>},
            {sparse_caller<4, 1, 3>, sparse_caller<4, 2, 3>, sparse_caller<4, 3, 3>, sparse_caller<4, 4, 3>, sparse_caller<4, 5, 3>},
            {sparse_caller<4, 1, 4>, sparse_caller<4, 2, 4>, sparse_caller<4, 3, 4>, sparse_caller<4, 4, 4>, sparse_caller<4, 5, 4>},
            {sparse_caller<4, 1, 5>, sparse_caller<4, 2, 5>, sparse_caller<4, 3, 5>, sparse_caller<4, 4, 5>, sparse_caller<4, 5, 5>}
        };

        bindTexture(&tex_If4, I);
        bindTexture(&tex_Jf4, J);

        funcs[patch.y - 1][patch.x - 1](I.rows, I.cols, prevPts, nextPts, status, err, ptcount,
            level, block, stream);
    }

#if defined(HAVE_TBB)
    void sparse1_multi(PtrStepSzf I, PtrStepSzf J, const float2* prevPts, float2* nextPts, uchar* status, float* err, int ptcount,
                 int level, dim3 block, dim3 patch, hipStream_t stream, int index)
    {
        typedef void (*func_t)(int rows, int cols, const float2* prevPts, float2* nextPts, uchar* status, float* err, int ptcount,
                               int level, dim3 block, hipStream_t stream);

        static const func_t funcs[5][5][5] =
        {
            { // index 0
                {sparse_caller_multi<1, 0, 1, 1>, sparse_caller_multi<1, 0, 2, 1>, sparse_caller_multi<1, 0, 3, 1>, sparse_caller_multi<1, 0, 4, 1>, sparse_caller_multi<1, 0, 5, 1>},
                {sparse_caller_multi<1, 0, 1, 2>, sparse_caller_multi<1, 0, 2, 2>, sparse_caller_multi<1, 0, 3, 2>, sparse_caller_multi<1, 0, 4, 2>, sparse_caller_multi<1, 0, 5, 2>},
                {sparse_caller_multi<1, 0, 1, 3>, sparse_caller_multi<1, 0, 2, 3>, sparse_caller_multi<1, 0, 3, 3>, sparse_caller_multi<1, 0, 4, 3>, sparse_caller_multi<1, 0, 5, 3>},
                {sparse_caller_multi<1, 0, 1, 4>, sparse_caller_multi<1, 0, 2, 4>, sparse_caller_multi<1, 0, 3, 4>, sparse_caller_multi<1, 0, 4, 4>, sparse_caller_multi<1, 0, 5, 4>},
                {sparse_caller_multi<1, 0, 1, 5>, sparse_caller_multi<1, 0, 2, 5>, sparse_caller_multi<1, 0, 3, 5>, sparse_caller_multi<1, 0, 4, 5>, sparse_caller_multi<1, 0, 5, 5>}
            },
            { // index 1
                {sparse_caller_multi<1, 1, 1, 1>, sparse_caller_multi<1, 1, 2, 1>, sparse_caller_multi<1, 1, 3, 1>, sparse_caller_multi<1, 1, 4, 1>, sparse_caller_multi<1, 1, 5, 1>},
                {sparse_caller_multi<1, 1, 1, 2>, sparse_caller_multi<1, 1, 2, 2>, sparse_caller_multi<1, 1, 3, 2>, sparse_caller_multi<1, 1, 4, 2>, sparse_caller_multi<1, 1, 5, 2>},
                {sparse_caller_multi<1, 1, 1, 3>, sparse_caller_multi<1, 1, 2, 3>, sparse_caller_multi<1, 1, 3, 3>, sparse_caller_multi<1, 1, 4, 3>, sparse_caller_multi<1, 1, 5, 3>},
                {sparse_caller_multi<1, 1, 1, 4>, sparse_caller_multi<1, 1, 2, 4>, sparse_caller_multi<1, 1, 3, 4>, sparse_caller_multi<1, 1, 4, 4>, sparse_caller_multi<1, 1, 5, 4>},
                {sparse_caller_multi<1, 1, 1, 5>, sparse_caller_multi<1, 1, 2, 5>, sparse_caller_multi<1, 1, 3, 5>, sparse_caller_multi<1, 1, 4, 5>, sparse_caller_multi<1, 1, 5, 5>}
            },
            { // index 2
                {sparse_caller_multi<1, 2, 1, 1>, sparse_caller_multi<1, 2, 2, 1>, sparse_caller_multi<1, 2, 3, 1>, sparse_caller_multi<1, 2, 4, 1>, sparse_caller_multi<1, 2, 5, 1>},
                {sparse_caller_multi<1, 2, 1, 2>, sparse_caller_multi<1, 2, 2, 2>, sparse_caller_multi<1, 2, 3, 2>, sparse_caller_multi<1, 2, 4, 2>, sparse_caller_multi<1, 2, 5, 2>},
                {sparse_caller_multi<1, 2, 1, 3>, sparse_caller_multi<1, 2, 2, 3>, sparse_caller_multi<1, 2, 3, 3>, sparse_caller_multi<1, 2, 4, 3>, sparse_caller_multi<1, 2, 5, 3>},
                {sparse_caller_multi<1, 2, 1, 4>, sparse_caller_multi<1, 2, 2, 4>, sparse_caller_multi<1, 2, 3, 4>, sparse_caller_multi<1, 2, 4, 4>, sparse_caller_multi<1, 2, 5, 4>},
                {sparse_caller_multi<1, 2, 1, 5>, sparse_caller_multi<1, 2, 2, 5>, sparse_caller_multi<1, 2, 3, 5>, sparse_caller_multi<1, 2, 4, 5>, sparse_caller_multi<1, 2, 5, 5>}
            },
            { // index 3
                {sparse_caller_multi<1, 3, 1, 1>, sparse_caller_multi<1, 3, 2, 1>, sparse_caller_multi<1, 3, 3, 1>, sparse_caller_multi<1, 3, 4, 1>, sparse_caller_multi<1, 3, 5, 1>},
                {sparse_caller_multi<1, 3, 1, 2>, sparse_caller_multi<1, 3, 2, 2>, sparse_caller_multi<1, 3, 3, 2>, sparse_caller_multi<1, 3, 4, 2>, sparse_caller_multi<1, 3, 5, 2>},
                {sparse_caller_multi<1, 3, 1, 3>, sparse_caller_multi<1, 3, 2, 3>, sparse_caller_multi<1, 3, 3, 3>, sparse_caller_multi<1, 3, 4, 3>, sparse_caller_multi<1, 3, 5, 3>},
                {sparse_caller_multi<1, 3, 1, 4>, sparse_caller_multi<1, 3, 2, 4>, sparse_caller_multi<1, 3, 3, 4>, sparse_caller_multi<1, 3, 4, 4>, sparse_caller_multi<1, 3, 5, 4>},
                {sparse_caller_multi<1, 3, 1, 5>, sparse_caller_multi<1, 3, 2, 5>, sparse_caller_multi<1, 3, 3, 5>, sparse_caller_multi<1, 3, 4, 5>, sparse_caller_multi<1, 3, 5, 5>}
            },
            { // index 4
                {sparse_caller_multi<1, 4, 1, 1>, sparse_caller_multi<1, 4, 2, 1>, sparse_caller_multi<1, 4, 3, 1>, sparse_caller_multi<1, 4, 4, 1>, sparse_caller_multi<1, 4, 5, 1>},
                {sparse_caller_multi<1, 4, 1, 2>, sparse_caller_multi<1, 4, 2, 2>, sparse_caller_multi<1, 4, 3, 2>, sparse_caller_multi<1, 4, 4, 2>, sparse_caller_multi<1, 4, 5, 2>},
                {sparse_caller_multi<1, 4, 1, 3>, sparse_caller_multi<1, 4, 2, 3>, sparse_caller_multi<1, 4, 3, 3>, sparse_caller_multi<1, 4, 4, 3>, sparse_caller_multi<1, 4, 5, 3>},
                {sparse_caller_multi<1, 4, 1, 4>, sparse_caller_multi<1, 4, 2, 4>, sparse_caller_multi<1, 4, 3, 4>, sparse_caller_multi<1, 4, 4, 4>, sparse_caller_multi<1, 4, 5, 4>},
                {sparse_caller_multi<1, 4, 1, 5>, sparse_caller_multi<1, 4, 2, 5>, sparse_caller_multi<1, 4, 3, 5>, sparse_caller_multi<1, 4, 4, 5>, sparse_caller_multi<1, 4, 5, 5>}
            }
        };

        switch(index)
        {
            case 0:
                bindTexture(&tex_If1_multi0, I);
                bindTexture(&tex_Jf1_multi0, J);
                break;
            case 1:
                bindTexture(&tex_If1_multi1, I);
                bindTexture(&tex_Jf1_multi1, J);
                break;
            case 2:
                bindTexture(&tex_If1_multi2, I);
                bindTexture(&tex_Jf1_multi2, J);
                break;
            case 3:
                bindTexture(&tex_If1_multi3, I);
                bindTexture(&tex_Jf1_multi3, J);
                break;
            case 4:
                bindTexture(&tex_If1_multi4, I);
                bindTexture(&tex_Jf1_multi4, J);
                break;
            default:
                CV_Error(CV_StsBadArg, "invalid execution line index");
                break;
        }

        funcs[index][patch.y - 1][patch.x - 1](I.rows, I.cols, prevPts, nextPts, status, err, ptcount,
            level, block, stream);
    }

    void sparse4_multi(PtrStepSz<float4> I, PtrStepSz<float4> J, const float2* prevPts, float2* nextPts, uchar* status, float* err, int ptcount,
                 int level, dim3 block, dim3 patch, hipStream_t stream, int index)
    {
        typedef void (*func_t)(int rows, int cols, const float2* prevPts, float2* nextPts, uchar* status, float* err, int ptcount,
                               int level, dim3 block, hipStream_t stream);

        static const func_t funcs[5][5][5] =
        {
            { // index 0
                {sparse_caller_multi<4, 0, 1, 1>, sparse_caller_multi<4, 0, 2, 1>, sparse_caller_multi<4, 0, 3, 1>, sparse_caller_multi<4, 0, 4, 1>, sparse_caller_multi<4, 0, 5, 1>},
                {sparse_caller_multi<4, 0, 1, 2>, sparse_caller_multi<4, 0, 2, 2>, sparse_caller_multi<4, 0, 3, 2>, sparse_caller_multi<4, 0, 4, 2>, sparse_caller_multi<4, 0, 5, 2>},
                {sparse_caller_multi<4, 0, 1, 3>, sparse_caller_multi<4, 0, 2, 3>, sparse_caller_multi<4, 0, 3, 3>, sparse_caller_multi<4, 0, 4, 3>, sparse_caller_multi<4, 0, 5, 3>},
                {sparse_caller_multi<4, 0, 1, 4>, sparse_caller_multi<4, 0, 2, 4>, sparse_caller_multi<4, 0, 3, 4>, sparse_caller_multi<4, 0, 4, 4>, sparse_caller_multi<4, 0, 5, 4>},
                {sparse_caller_multi<4, 0, 1, 5>, sparse_caller_multi<4, 0, 2, 5>, sparse_caller_multi<4, 0, 3, 5>, sparse_caller_multi<4, 0, 4, 5>, sparse_caller_multi<4, 0, 5, 5>}
            },
            { // index 1
                {sparse_caller_multi<4, 1, 1, 1>, sparse_caller_multi<4, 1, 2, 1>, sparse_caller_multi<4, 1, 3, 1>, sparse_caller_multi<4, 1, 4, 1>, sparse_caller_multi<4, 1, 5, 1>},
                {sparse_caller_multi<4, 1, 1, 2>, sparse_caller_multi<4, 1, 2, 2>, sparse_caller_multi<4, 1, 3, 2>, sparse_caller_multi<4, 1, 4, 2>, sparse_caller_multi<4, 1, 5, 2>},
                {sparse_caller_multi<4, 1, 1, 3>, sparse_caller_multi<4, 1, 2, 3>, sparse_caller_multi<4, 1, 3, 3>, sparse_caller_multi<4, 1, 4, 3>, sparse_caller_multi<4, 1, 5, 3>},
                {sparse_caller_multi<4, 1, 1, 4>, sparse_caller_multi<4, 1, 2, 4>, sparse_caller_multi<4, 1, 3, 4>, sparse_caller_multi<4, 1, 4, 4>, sparse_caller_multi<4, 1, 5, 4>},
                {sparse_caller_multi<4, 1, 1, 5>, sparse_caller_multi<4, 1, 2, 5>, sparse_caller_multi<4, 1, 3, 5>, sparse_caller_multi<4, 1, 4, 5>, sparse_caller_multi<4, 1, 5, 5>}
            },
            { // index 2
                {sparse_caller_multi<4, 2, 1, 1>, sparse_caller_multi<4, 2, 2, 1>, sparse_caller_multi<4, 2, 3, 1>, sparse_caller_multi<4, 2, 4, 1>, sparse_caller_multi<4, 2, 5, 1>},
                {sparse_caller_multi<4, 2, 1, 2>, sparse_caller_multi<4, 2, 2, 2>, sparse_caller_multi<4, 2, 3, 2>, sparse_caller_multi<4, 2, 4, 2>, sparse_caller_multi<4, 2, 5, 2>},
                {sparse_caller_multi<4, 2, 1, 3>, sparse_caller_multi<4, 2, 2, 3>, sparse_caller_multi<4, 2, 3, 3>, sparse_caller_multi<4, 2, 4, 3>, sparse_caller_multi<4, 2, 5, 3>},
                {sparse_caller_multi<4, 2, 1, 4>, sparse_caller_multi<4, 2, 2, 4>, sparse_caller_multi<4, 2, 3, 4>, sparse_caller_multi<4, 2, 4, 4>, sparse_caller_multi<4, 2, 5, 4>},
                {sparse_caller_multi<4, 2, 1, 5>, sparse_caller_multi<4, 2, 2, 5>, sparse_caller_multi<4, 2, 3, 5>, sparse_caller_multi<4, 2, 4, 5>, sparse_caller_multi<4, 2, 5, 5>}
            },
            { // index 3
                {sparse_caller_multi<4, 3, 1, 1>, sparse_caller_multi<4, 3, 2, 1>, sparse_caller_multi<4, 3, 3, 1>, sparse_caller_multi<4, 3, 4, 1>, sparse_caller_multi<4, 3, 5, 1>},
                {sparse_caller_multi<4, 3, 1, 2>, sparse_caller_multi<4, 3, 2, 2>, sparse_caller_multi<4, 3, 3, 2>, sparse_caller_multi<4, 3, 4, 2>, sparse_caller_multi<4, 3, 5, 2>},
                {sparse_caller_multi<4, 3, 1, 3>, sparse_caller_multi<4, 3, 2, 3>, sparse_caller_multi<4, 3, 3, 3>, sparse_caller_multi<4, 3, 4, 3>, sparse_caller_multi<4, 3, 5, 3>},
                {sparse_caller_multi<4, 3, 1, 4>, sparse_caller_multi<4, 3, 2, 4>, sparse_caller_multi<4, 3, 3, 4>, sparse_caller_multi<4, 3, 4, 4>, sparse_caller_multi<4, 3, 5, 4>},
                {sparse_caller_multi<4, 3, 1, 5>, sparse_caller_multi<4, 3, 2, 5>, sparse_caller_multi<4, 3, 3, 5>, sparse_caller_multi<4, 3, 4, 5>, sparse_caller_multi<4, 3, 5, 5>}
            },
            { // index 4
                {sparse_caller_multi<4, 4, 1, 1>, sparse_caller_multi<4, 4, 2, 1>, sparse_caller_multi<4, 4, 3, 1>, sparse_caller_multi<4, 4, 4, 1>, sparse_caller_multi<4, 4, 5, 1>},
                {sparse_caller_multi<4, 4, 1, 2>, sparse_caller_multi<4, 4, 2, 2>, sparse_caller_multi<4, 4, 3, 2>, sparse_caller_multi<4, 4, 4, 2>, sparse_caller_multi<4, 4, 5, 2>},
                {sparse_caller_multi<4, 4, 1, 3>, sparse_caller_multi<4, 4, 2, 3>, sparse_caller_multi<4, 4, 3, 3>, sparse_caller_multi<4, 4, 4, 3>, sparse_caller_multi<4, 4, 5, 3>},
                {sparse_caller_multi<4, 4, 1, 4>, sparse_caller_multi<4, 4, 2, 4>, sparse_caller_multi<4, 4, 3, 4>, sparse_caller_multi<4, 4, 4, 4>, sparse_caller_multi<4, 4, 5, 4>},
                {sparse_caller_multi<4, 4, 1, 5>, sparse_caller_multi<4, 4, 2, 5>, sparse_caller_multi<4, 4, 3, 5>, sparse_caller_multi<4, 4, 4, 5>, sparse_caller_multi<4, 4, 5, 5>}
            }
        };

        switch(index)
        {
            case 0:
                bindTexture(&tex_If4_multi0, I);
                bindTexture(&tex_Jf4_multi0, J);
                break;
            case 1:
                bindTexture(&tex_If4_multi1, I);
                bindTexture(&tex_Jf4_multi1, J);
                break;
            case 2:
                bindTexture(&tex_If4_multi2, I);
                bindTexture(&tex_Jf4_multi2, J);
                break;
            case 3:
                bindTexture(&tex_If4_multi3, I);
                bindTexture(&tex_Jf4_multi3, J);
                break;
            case 4:
                bindTexture(&tex_If4_multi4, I);
                bindTexture(&tex_Jf4_multi4, J);
                break;
            default:
                CV_Error(CV_StsBadArg, "invalid execution line index");
                break;
        }

        funcs[index][patch.y - 1][patch.x - 1](I.rows, I.cols, prevPts, nextPts, status, err, ptcount,
            level, block, stream);
    }

#endif // defined(HAVE_TBB)

    void dense(PtrStepSzb I, PtrStepSzf J, PtrStepSzf u, PtrStepSzf v, PtrStepSzf prevU, PtrStepSzf prevV, PtrStepSzf err, int2 winSize, hipStream_t stream)
    {
        dim3 block(16, 16);
        dim3 grid(divUp(I.cols, block.x), divUp(I.rows, block.y));

        bindTexture(&tex_Ib, I);
        bindTexture(&tex_Jf, J);

        int2 halfWin = make_int2((winSize.x - 1) / 2, (winSize.y - 1) / 2);
        const int patchWidth  = block.x + 2 * halfWin.x;
        const int patchHeight = block.y + 2 * halfWin.y;
        size_t smem_size = 3 * patchWidth * patchHeight * sizeof(int);

        if (err.data)
        {
            denseKernel<true><<<grid, block, smem_size, stream>>>(u, v, prevU, prevV, err, I.rows, I.cols);
            cudaSafeCall( hipGetLastError() );
        }
        else
        {
            denseKernel<false><<<grid, block, smem_size, stream>>>(u, v, prevU, prevV, PtrStepf(), I.rows, I.cols);
            cudaSafeCall( hipGetLastError() );
        }

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }
}

#endif /* CUDA_DISABLER */
